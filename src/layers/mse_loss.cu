#include "hip/hip_runtime.h"
#include "include/cnn_library/layers/loss.h"
#include <stdexcept>
#include <vector>

class MSE_Loss : public Loss {
  public:
    MSE_Loss(size_t batch_size) {
        this->layer_name = "MSE_Loss";
        this->batch_size = batch_size;

        this->host_forward_buffer = new float[1];
        this->host_backward_buffer = new float[batch_size];
    }

    ~MSE_Loss() {
        delete[] host_backward_buffer;
        delete[] host_forward_buffer;
        if (device) {
            hipFree(d_loss);
            hipFree(device_backward_buffer);
        }
    }

    float *forward(const float *pred) {
        float loss;
        if (this->device) {
            hipMemset(d_loss, 0, sizeof(float));
            forward_kernel<<<1, 256>>>(pred, this->target, d_loss, batch_size);
            hipMemcpy(&loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
        } else {
            loss = forward_CPU(pred, this->target);
        }
        host_forward_buffer[0] = loss;
        return host_forward_buffer;
    }

    float *backward(float *pred) {
        if (this->device) {
            backward_kernel(device_backward_buffer, pred, target, batch_size);
            return device_backward_buffer;
        } else {
            backward_CPU(host_backward_buffer, pred, target);
            return host_backward_buffer;
        }
    }

    void setDevice(int device) override {
        this->device = device;
        if (device) {
            hipMalloc(&d_loss, sizeof(float));
            hipMalloc(&device_backward_buffer, sizeof(float) * batch_size);
        }
    }
    void setTarget(float *target) override { this->target = target; }

    float forward_CPU(const float *pred, float *target) {
        int n = batch_size;

        float loss = 0.0;
        for (auto i = 0; i < n; i++) {
            loss += pow(pred[i] - target[i], 2);
        }
        loss /= static_cast<float>(n);
        return loss;
    }

    // TODO: use reduction to avoid atomic adds
    __global__ void forward_kernel(const float *pred, float *target, float *loss, int n) {
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        if (idx < n) {
            float diff = pred[idx] - target[idx];
            atomicAdd(loss, diff * diff / n);
        }
    }

    void backward_CPU(float *grad_output, float *pred, float *target) {
        int n = batch_size;

        for (auto i = 0; i < n; i++) {
            grad_output[i] = 2.0 * (pred[i] - target[i]) / n;
        }
    }

    __global__ void backward_kernel(float *grad_output, float *pred, float *target, int n) {
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        if (idx < n) {
            grad_output[idx] = 2.0 * (pred[idx] - target[idx]) / n;
        }
    }

  private:
    float *d_loss;
};