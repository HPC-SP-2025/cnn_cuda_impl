#include "hip/hip_runtime.h"
# include "../../include/cnn_library/layers/linear.h"
# include <iostream>

// Constructor
Linear::Linear(size_t input_size, size_t output_size, size_t batch_size) {
    this->layer_name = "Linear";
    this->input_size = input_size;
    this->output_size = output_size;
    this->batch_size = batch_size;

    // Allocate memory for weights and biases
    size_t weights_size = input_size * output_size;
    host_weights = new float[weights_size];
    host_biases = new float[output_size];

    // Allocate memory for gradients
    host_grad_weights = new float[weights_size];
    host_grad_biases = new float[output_size];

    // Allocate buffers for forward and backward passes
    host_forward_buffer = new float[output_size * batch_size];
    host_backward_buffer = new float[input_size * batch_size];

    // Initialize device pointers to nullptr
    device_weights = nullptr;
    device_biases = nullptr;
    device_grad_weights = nullptr;
    device_grad_biases = nullptr;
    device_forward_buffer = nullptr;
    device_backward_buffer = nullptr;

    // Allocate cached input
    cached_input = new float[input_size * batch_size];

    // Initialize weights and biases
    initializeWeights();
    initializeBiases();

    // TODO: Is this necessary?
    // Initialize gradients to zero
    std::memset(host_grad_weights, 0, weights_size * sizeof(float));
    std::memset(host_grad_biases, 0, output_size * sizeof(float));

    std::cout << "Linear layer created with input size: " << input_size 
              << ", output size: " << output_size 
              << ", batch size: " << batch_size << std::endl;
}

// Destructor
Linear::~Linear() {
    // Free host memory
    delete[] host_weights;
    delete[] host_biases;
    delete[] host_grad_weights;
    delete[] host_grad_biases;
    delete[] host_forward_buffer;
    delete[] host_backward_buffer;
    delete[] cached_input;

    if (device) {
        // TODO: Free CUDA memory
    }

    std::cout << "Linear layer destroyed" << std::endl;
}

// Forward
void Linear::forward(float* input, float* output) {
    // Cache input for backward pass
    std::memcpy(cached_input, input, input_size * batch_size * sizeof(float));

    if(!device) {
        // CPU forward pass
        forwardCPU(input, host_forward_buffer);
    }
    else {
        // TODO: GPU forward pass

    }
}


// CPU forward pass
void Linear::forwardCPU(float* input, float* output) {
    // Initialize output with biases
    for (size_t b = 0; b < batch_size; b++) {
        for (size_t o = 0; o < output_size; o++) {
            output[b * output_size + o] = host_biases[o];
        }
    }

    // Matrix multiplication: output += input * weights
    for (size_t b = 0; b < batch_size; b++) {
        for (size_t i = 0; i < input_size; i++) {
            for (size_t o = 0; o < output_size; o++) {
                output[b * output_size + o] += input[b * input_size + i] * host_weights[i * output_size + o];
            }
        }
    }
}

// Backward
void Linear::backward(float* grad_input, float* grad_output) {
    if(!device) {
        // CPU implementation
        backwardCPU(grad_input, grad_output);
    } else {
        // TODO: GPU backward pass
    }
}

// CPU backward pass
void Linear::backwardCPU(float* grad_input, float* grad_output) {
    // Initialize grad_output with zeros
    std::memset(grad_output, 0, input_size * batch_size * sizeof(float));

    // Compute di
    for (size_t b = 0; b < batch_size; b++) {
        for (size_t i = 0; i < input_size; i++) {
            for (size_t o = 0; o < output_size; o++) {
                grad_output[b * input_size + i] += grad_input[b * output_size + o] * host_weights[i * output_size + o];
            }
        }
    }

    // Compute dw
    for (size_t b = 0; b < batch_size; b++) {
        for (size_t i = 0; i < input_size; i++) {
            for (size_t o = 0; o < output_size; o++) {
                host_grad_weights[i * output_size + o] += 
                    cached_input[b * input_size + i] * grad_input[b * output_size + o];
            }
        }
    }

    // Compute db
    for (size_t b = 0; b < batch_size; b++) {
        for (size_t o = 0; o < output_size; o++) {
            host_grad_biases[o] += grad_input[b * output_size + o];
        }
    }
}

// Set device (0 - CPU, 1 - GPU)
// TODO: Change `device` to enum
void Linear::setDevice(int device) {
    this->device = device;

    if (device) {
        // TODO: Allocate and copy memory to GPU
    }
}

// Update weights with gradients
void Linear::updateParameters(float learning_rate) {
    if (!device) {
        // Update weights on CPU
        for (size_t i = 0; i < input_size * output_size; i++) {
            host_weights[i] -= learning_rate * host_grad_weights[i];
        }
        
        // Update biases on CPU
        for (size_t i = 0; i < output_size; i++) {
            host_biases[i] -= learning_rate * host_grad_biases[i];
        }

        // Reset gradients
        std::memset(host_grad_weights, 0, input_size * output_size * sizeof(float));
        std::memset(host_grad_biases, 0, output_size * sizeof(float));
    } else {
        // TODO: GPU impl
    }
}

// Initialize weights with Xavier initialization
void Linear::initializeWeights() {
    std::random_device rd;
    std::mt19937 gen(rd());

    // Xavier init: stddev = sqrt(2 / (input_size + output_size))
    float stddev = std::sqrt(2.0f / (input_size | output_size));
    std::normal_distribution<float> d(0.0f, stddev);

    // Initialize weights
    for (size_t i = 0; i < input_size * output_size; i++) {
        host_weights[i] = d(gen);
    }
}

// Initialize biases to zero
void Linear::initializeBiases() {
    std::memset(host_biases, 0, output_size * sizeof(float));
}

// Get input size
size_t Linear::getInputSize() {
    return input_size;
}

// Get output size
size_t Linear::getOutputSize() {
    return output_size;
}

// Get number of parameters (weights + biases)
size_t Linear::numParams() {
    return (input_size * output_size) + output_size;
}

// Get layer name
std::string Linear::getLayerName() {
    return layer_name;
}

// Get device
int Linear::getDevice() {
    return device;
}

void Linear::forwardGPU(float* input, float* output) {

}

void Linear::backwardGPU(float* grad_input, float* grad_output) {

}
