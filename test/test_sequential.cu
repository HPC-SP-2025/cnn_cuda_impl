#include <iostream>
#include "../include/cnn_library/nn/sequential.h"
#include "../include/cnn_library/layers/relu.h"
#include "../include/cnn_library/layers/softmax.h"
#include "../include/cnn_library/layers/cross_entropy_loss.h"
#include "../include/cnn_library/layers/linear.h"



using namespace std;

// Function to create and return a random array with random positive or negative numbers
#include <cstdlib>
#include <ctime>

float* createAndReturnRandomArray(unsigned int size) 
{
    const float constantValue = 5.0f; // Constant value to fill the array
    float* constantArray = new float[size];
    for (unsigned int i = 0; i < size; ++i) {
        constantArray[i] = constantValue;
    }
    return constantArray;
}

int main(){

    int size = 5;    


    // Create a Sequential model
    Sequential* model = new Sequential(size, size); // Example input size of 784 (28x28 image flattened) and output size of 10 (number of classes)

    cout << "Model created with input size: " << model->getInputSize() << " and output size: " << model->getOutputSize() << endl;
    model->addLayer(new Linear(5, 5, 1));
    model->addLayer(new ReLU(5, 5, 1)); // Example layer with input size 784, output size 10, and batch size 32
    model->addLayer(new Linear(5, 5, 1)); // Example layer with input size 784, output size 10, and batch size 32
    // model->addLayer(new Softmax(5, 1)); // Example layer with input size 784, output size 10, and batch size 32
    model->loadModel("/home/MORGRIDGE/akazi/HPC_Assignments/Final_Project/CNN_Implementation_on_CUDA/test/model_weights.txt");

    model->summary();
    
    // LOSS FUNCTION
    // Cross_Entropy_Loss* loss_fn = new Cross_Entropy_Loss(5, 1); // Example layer with input size 784, output size 10, and batch size 32
    // loss_fn->setTarget(new float[1]{4}); // Set the target for the loss function
    // loss_fn->setDevice(1); // Set the device to CPU (0 for CPU, 1 for GPU)

    
    
    
//     for(int j = 0; j < 10; j++)
//     {
//         cout << "Iteration: " << j << endl;

//         // Input Image
//         float* h_arr = createAndReturnRandomArray(size);
//         float* arr;

//         // Allocate memory on the device
//         hipMalloc((void**)&arr, size * sizeof(float));

//         // Copy the array from host to device
//         hipMemcpy(arr, h_arr, size * sizeof(float), hipMemcpyHostToDevice);


        


//         // Grouth Truth
//         // int targetIndex = 0;
//         // for (int i = 1; i < size; ++i) 
//         // {
//         //     if (arr[i] > arr[targetIndex]) 
//         //     {
//         //         targetIndex = i;
//         //     }
//         // }

//         // loss_fn->setTarget(new float[1]{static_cast<float>(targetIndex)}); // Set the target for the loss function
//         // float* arr = new float[size]{-1.0, -.05, 0, 0, 0.8}; // Example input array
//         cout << "Image Pointer: " << arr << endl;

//         // Add layers to the model
//         float* output = model->forward(arr);
//         // float* loss_value = loss_fn->forward(output);
//         // cout << "Loss Value: " << *loss_value << endl;

//         // Move the output from device to host
//         float* h_output = new float[model->getOutputSize()];
//         hipMemcpy(h_output, output, model->getOutputSize() * sizeof(float), hipMemcpyDeviceToHost);

  
//         // Use h_output for further processing if needed
        

//         // // Print the array
//         // cout << "";
//         // for (int i = 0; i < size; ++i) {
//         //     cout << arr[i] << " ";
//         //     // printf("%f ", arr[i]);
//         // }
//         // cout << endl;


//         cout << endl;
//         cout << "";
//         for (int i = 0; i < model->getOutputSize(); ++i) {
//             cout << h_output[i] << " ";
//             // printf("%f ", output[i]);
//         }
//         cout << endl;

//  }
    

    return 0;
}


